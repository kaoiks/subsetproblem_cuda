#include <iostream>
#include <hip/hip_runtime.h>

#include <vector>
#include <chrono>


__global__ void subsetSumProblemCuda(int n, int* dataGlobal, int* solutionsFoundMain) {
   int threadId = threadIdx.x;
   extern __shared__ int data[];
   __shared__ int solutionsFound;
   solutionsFound = 0;

   for (int i = threadId; i < n; i += blockDim.x) {
       data[i] = dataGlobal[i];
   }
   int sum = 0;
   for (long long i = threadId + 1; i < (1ULL << n); i+= blockDim.x) {
       int sub_sum = 0;
       for (int j = 0; j < n; ++j) {
           if ((1ULL << j) & i) {
               sub_sum += data[j];
           }
       }
       if (sub_sum == sum) {
           atomicAdd(&solutionsFound, 1);
       }
       
   }
   __syncthreads();
   if (threadId == 0) {
       *solutionsFoundMain = solutionsFound;
   }
}


int subsetSumProblemSequential(int* data, int n) {
   int solutionsFound = 0;
   int sum = 0;

   for (long long i = 1; i < (1ULL << n); ++i) {
       int sub_sum = 0;
       for (int j = 0; j < n; ++j) {
           if ((1ULL << j) & i) {
               sub_sum += data[j];
           }
       }
       if (sub_sum == sum) {
           ++solutionsFound;
       }
   }
   return solutionsFound;
}



int generateSubset(int* data, int SIZE) {
   std::srand(std::time(nullptr));

   for (int i = 0; i < SIZE; i++) {
       int num = std::rand() % 201 - 100;

       data[i] = num;
   }

   // Print the generated array
   std::cout << "Generated Array:" << std::endl;
   for (int i = 0; i < SIZE; i++) {
       std::cout << data[i] << " ";
   }
   std::cout << std::endl;

   return 0;
}

int main() {
   
   int sum = 0;
   
   #define SIZE 30

   int data[SIZE];
   generateSubset(data, SIZE);
   /*int data[] = { 1, 2 , -2};*/
   int n = sizeof(data)/ sizeof(data[0]);
   int* dataDevice;
   int* hasFound;

   std::cout << "CUDA" << std::endl;
   auto start_time = std::chrono::high_resolution_clock::now();
   hipMalloc(&dataDevice, n*sizeof(int));
   hipMalloc(&hasFound, sizeof(int));
   hipMemcpy(dataDevice, data, n * sizeof(int), hipMemcpyHostToDevice);
   subsetSumProblemCuda <<<1, 1024, n * sizeof(int)>>>(n, dataDevice, hasFound);

   hipDeviceSynchronize();
   int hasFoundHost;
   hipMemcpy(&hasFoundHost, hasFound, sizeof(int), hipMemcpyDeviceToHost);

   hipFree(dataDevice);
   hipFree(hasFound);


   if (hasFoundHost) {
       std::cout << "Found a subset with given sum" << std::endl;
       std::cout << hasFoundHost << std::endl;;
   }
   else
       std::cout << "No subset with given sum" << std::endl;

   auto count_duration = std::chrono::high_resolution_clock::now() - start_time;
   double duration_seconds = std::chrono::duration<double>(count_duration).count();
   std::cout << duration_seconds << std::endl;


   std::cout << "Sequential" << std::endl;
   start_time = std::chrono::high_resolution_clock::now();
   int foundResultsSequential = subsetSumProblemSequential(data, n);
   if (foundResultsSequential) {
       std::cout << "Found a subset with given sum" << std::endl;
       std::cout << foundResultsSequential << std::endl;;
   }
   else
       std::cout << "No subset with given sum" << std::endl;

   count_duration = std::chrono::high_resolution_clock::now() - start_time;
   duration_seconds = std::chrono::duration<double>(count_duration).count();
   std::cout << duration_seconds << std::endl;

   return 0;
}