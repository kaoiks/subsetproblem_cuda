﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


int subsetSumProblemSequential(int* data, int n) {
    int solutionsFound = 0;
    int sum = 0;

    for (long long i = 1; i < (1ULL << n); ++i) {
        int sub_sum = 0;
        for (int j = 0; j < n; ++j) {
            if ((1ULL << j) & i) {
                sub_sum += data[j];
            }
        }
        if (sub_sum == sum) {
            ++solutionsFound;
        }
    }
    return solutionsFound;
}

int generateSubset(int* data, int SIZE) {
    std::srand(std::time(nullptr));

    for (int i = 0; i < SIZE; i++) {
        int num = std::rand() % 201 - 100;

        data[i] = num;
    }

    // Print the generated array
    std::cout << "Generated Array:" << std::endl;
    for (int i = 0; i < SIZE; i++) {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}

int main() {

    int sum = 0;

    #define SIZE 29

    int data[SIZE];
    generateSubset(data, SIZE);
    /*int data[] = { 1, 2 , -2};*/
    int n = sizeof(data) / sizeof(data[0]);
    int* dataDevice;
    int* hasFound;

    
    auto start_time = std::chrono::high_resolution_clock::now();


    std::cout << "Sequential" << std::endl;
    int foundResultsSequential = subsetSumProblemSequential(data, n);
    if (foundResultsSequential) {
        std::cout << "Found a subset with given sum" << std::endl;
        std::cout << foundResultsSequential << std::endl;;
    }
    else
        std::cout << "No subset with given sum" << std::endl;

    auto count_duration = std::chrono::high_resolution_clock::now() - start_time;
    double duration_seconds = std::chrono::duration<double>(count_duration).count();
    std::cout << duration_seconds << std::endl;

    return 0;
}